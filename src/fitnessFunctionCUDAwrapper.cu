#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
//#include <cuda_runtime.h>
#define DT double
#define TEM_CELLS 125
#define OPTIMIZED_KERNEL 1
void Check_CUDA_Error(const char *message)
    {
       hipError_t error = hipGetLastError();
       if(error!=hipSuccess) {
          fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
          exit(-1);
       }
    }


__global__ void fitnessKernel(
				int slides,int cols,int rows,int tem_cells,
                int tem_slides,int tem_cols,int tem_rows,
                int *tem_cells_slides_d,int *tem_cells_cols_d,
				int *tem_cells_rows_d,
			    int npatterns, int *freq_realization_d,
				int *patternArray_d,
			    int *realization_matrix_d)
{

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int tidz=threadIdx.z;
    int bidx=blockIdx.x;
    int bidy=blockIdx.y;
    int bidz=blockIdx.z;
    int bdimx=blockDim.x;
    int bdimy=blockDim.y;
    int bdimz=blockDim.z;
    int idx = bidx*bdimx + tidx;
    int idy = bidy*bdimy + tidy;
    int idz = bidz*bdimz + tidz;
//    int threadIdGPU = tidx + bdimx*tidy + bdimx*bdimy*tidz;
    int icell,pos;
    int local_pattern[TEM_CELLS];
    int minn,maxx,isFound,value;

    int mult_pos_tem_cells;
    pos = 0;
    minn=0;
    maxx=npatterns-1;
    isFound=0;

    if(idx < rows && idy < cols && idz < slides){


        for (icell = 0; icell < tem_cells; icell += 5){
            local_pattern[icell] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell]) +
                           (idy - 1 + tem_cells_cols_d[icell])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell])
                          *(rows * cols)];
            local_pattern[icell + 1] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 1]) +
                           (idy - 1 + tem_cells_cols_d[icell + 1])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 1])
                          *(rows * cols)];
            local_pattern[icell + 2] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 2]) +
                           (idy - 1 + tem_cells_cols_d[icell + 2])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 2])
                          *(rows * cols)];
            local_pattern[icell + 3] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 3]) +
                           (idy - 1 + tem_cells_cols_d[icell + 3])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 3])
                          *(rows * cols)];
            local_pattern[icell + 4] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 4]) +
                           (idy - 1 + tem_cells_cols_d[icell + 4])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 4])
                          *(rows * cols)];
        }

        while(minn<=maxx & isFound == 0){

//            pos = (int)(((float)(minn+maxx))*0.5);
            pos = ((minn+maxx)/2);
            mult_pos_tem_cells = pos*tem_cells;


            value = 0;
            icell = 0;

            while ( value == 0 & icell < tem_cells){
                if (     patternArray_d[icell + mult_pos_tem_cells] > local_pattern[icell] ){
                    value = 1;
                    maxx = pos - 1;
                }
                else if ( patternArray_d[icell + mult_pos_tem_cells] < local_pattern[icell] ){
                    value = 1;
                    minn = pos + 1;
                }

                icell = icell + 1;
            }


            if (value == 0){
                atomicSub(&freq_realization_d[pos],1);
                isFound = 1;
            }
        }
    }



}
__global__ void fitnessKernelOptimized(
				int slides,int cols,int rows,int tem_cells,
                int tem_slides,int tem_cols,int tem_rows,
                int *tem_cells_slides_d,int *tem_cells_cols_d,
				int *tem_cells_rows_d,
			    int npatterns, int *freq_realization_d,
				int *patternArray_d,
			    int *realization_matrix_d)
{

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int tidz=threadIdx.z;
    int bidx=blockIdx.x;
    int bidy=blockIdx.y;
    int bidz=blockIdx.z;
    int bdimx=blockDim.x;
    int bdimy=blockDim.y;
    int bdimz=blockDim.z;
    int idx = bidx*bdimx + tidx;
    int idy = bidy*bdimy + tidy;
    int idz = bidz*bdimz + tidz;
//    int threadIdGPU = tidx + bdimx*tidy + bdimx*bdimy*tidz;
    int icell,pos;
    int local_pattern[TEM_CELLS];
    int minn,maxx,isFound,value;
    int l0,l1,l2,l3,l4,l5,l6,l7,l8,l9;
    int l10,l11,l12,l13,l14,l15,l16,l17,l18,l19;
    int l20,l21,l22,l23,l24,l25,l26,l27,l28,l29;
    int l30,l31,l32,l33,l34,l35,l36,l37,l38,l39;
    int mult_pos_tem_cells;
    pos = 0;
    minn=0;
    maxx=npatterns-1;
    isFound=0;

    if(idx < rows && idy < cols && idz < slides){


        for (icell = 0; icell < tem_cells; icell += 5){
            local_pattern[icell] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell]) +
                           (idy - 1 + tem_cells_cols_d[icell])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell])
                          *(rows * cols)];
            local_pattern[icell + 1] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 1]) +
                           (idy - 1 + tem_cells_cols_d[icell + 1])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 1])
                          *(rows * cols)];
            local_pattern[icell + 2] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 2]) +
                           (idy - 1 + tem_cells_cols_d[icell + 2])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 2])
                          *(rows * cols)];
            local_pattern[icell + 3] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 3]) +
                           (idy - 1 + tem_cells_cols_d[icell + 3])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 3])
                          *(rows * cols)];
            local_pattern[icell + 4] = realization_matrix_d[(idx - 1 + tem_cells_rows_d[icell + 4]) +
                           (idy - 1 + tem_cells_cols_d[icell + 4])*(rows)
                          +(idz - 1 + tem_cells_slides_d[icell + 4])
                          *(rows * cols)];
        }
        l0 = local_pattern[0];
        l1 = local_pattern[1];
        l2 = local_pattern[2];
        l3 = local_pattern[3];
        l4 = local_pattern[4];
        l5 = local_pattern[5];
        l6 = local_pattern[6];
        l7 = local_pattern[7];
        l8 = local_pattern[8];
        l9 = local_pattern[9];
        l10 = local_pattern[10];
        l11 = local_pattern[11];
        l12 = local_pattern[12];
        l13 = local_pattern[13];
        l14 = local_pattern[14];
        l15 = local_pattern[15];
        l16 = local_pattern[16];
        l17 = local_pattern[17];
        l18 = local_pattern[18];
        l19 = local_pattern[19];
        l20 = local_pattern[20];
        l21 = local_pattern[21];
        l22 = local_pattern[22];
        l23 = local_pattern[23];
        l24 = local_pattern[24];
        l25 = local_pattern[25];
        l26 = local_pattern[26];
        l27 = local_pattern[27];
        l28 = local_pattern[28];
        l29 = local_pattern[29];
        l30 = local_pattern[30];
        l31 = local_pattern[31];
        l32 = local_pattern[32];
        l33 = local_pattern[33];
        l34 = local_pattern[34];
        l35 = local_pattern[35];
        l36 = local_pattern[36];
        l37 = local_pattern[37];
        l38 = local_pattern[38];
        l39 = local_pattern[39];

        while(minn<=maxx & isFound == 0){

//            pos = (int)(((float)(minn+maxx))*0.5);
            pos = ((minn+maxx)/2);
//            pos = (minn+maxx)<<2;

            value = 0;
            icell = 0;


            mult_pos_tem_cells = pos*tem_cells;

            while(true){
                value=patternArray_d[0 + mult_pos_tem_cells] - l0;//local_pattern[0];
                if(value!=0)break;
                value=patternArray_d[1 + mult_pos_tem_cells] - l1;//local_pattern[1];
                if(value!=0)break;
                value=patternArray_d[2 + mult_pos_tem_cells] - l2;//local_pattern[2];
                if(value!=0)break;
                value=patternArray_d[3 + mult_pos_tem_cells] - l3;//local_pattern[3];
                if(value!=0)break;
                value=patternArray_d[4 + mult_pos_tem_cells] - l4;//local_pattern[4];
                if(value!=0)break;
                value=patternArray_d[5 + mult_pos_tem_cells] - l5;//local_pattern[5];
                if(value!=0)break;
                value=patternArray_d[6 + mult_pos_tem_cells] - l6;//local_pattern[6];
                if(value!=0)break;
                value=patternArray_d[7 + mult_pos_tem_cells] - l7;//local_pattern[7];
                if(value!=0)break;
                value=patternArray_d[8 + mult_pos_tem_cells] - l8;//local_pattern[8];
                if(value!=0)break;
                value=patternArray_d[9 + mult_pos_tem_cells] - l9;//local_pattern[9];
                if(value!=0)break;
                value=patternArray_d[10 + mult_pos_tem_cells] - l10;//local_pattern[10];
                if(value!=0)break;
                value=patternArray_d[11 + mult_pos_tem_cells] - l11;//local_pattern[11];
                if(value!=0)break;
                value=patternArray_d[12 + mult_pos_tem_cells] - l12;//local_pattern[12];
                if(value!=0)break;
                value=patternArray_d[13 + mult_pos_tem_cells] - l13;//local_pattern[13];
                if(value!=0)break;
                value=patternArray_d[14 + mult_pos_tem_cells] - l14;//local_pattern[14];
                if(value!=0)break;
                value=patternArray_d[15 + mult_pos_tem_cells] - l15;//local_pattern[15];
                if(value!=0)break;
                value=patternArray_d[16 + mult_pos_tem_cells] - l16;//local_pattern[16];
                if(value!=0)break;
                value=patternArray_d[17 + mult_pos_tem_cells] - l17;//local_pattern[17];
                if(value!=0)break;
                value=patternArray_d[18 + mult_pos_tem_cells] - l18;//local_pattern[18];
                if(value!=0)break;
                value=patternArray_d[19 + mult_pos_tem_cells] - l19;//local_pattern[19];
                if(value!=0)break;
                value=patternArray_d[20 + mult_pos_tem_cells] - l20;//local_pattern[20];
                if(value!=0)break;
                value=patternArray_d[21 + mult_pos_tem_cells] - l21;//local_pattern[21];
                if(value!=0)break;
                value=patternArray_d[22 + mult_pos_tem_cells] - l22;//local_pattern[22];
                if(value!=0)break;
                value=patternArray_d[23 + mult_pos_tem_cells] - l23;//local_pattern[23];
                if(value!=0)break;
                value=patternArray_d[24 + mult_pos_tem_cells] - l24;//local_pattern[24];
                if(value!=0)break;
                value=patternArray_d[25 + mult_pos_tem_cells] - l25;//local_pattern[25];
                if(value!=0)break;
                value=patternArray_d[26 + mult_pos_tem_cells] - l26;//local_pattern[26];
                if(value!=0)break;
                value=patternArray_d[27 + mult_pos_tem_cells] - l27;//local_pattern[27];
                if(value!=0)break;
                value=patternArray_d[28 + mult_pos_tem_cells] - l28;//local_pattern[28];
                if(value!=0)break;
                value=patternArray_d[29 + mult_pos_tem_cells] - l29;//local_pattern[29];
                if(value!=0)break;
                value=patternArray_d[30 + mult_pos_tem_cells] - l30;//local_pattern[30];
                if(value!=0)break;
                value=patternArray_d[31 + mult_pos_tem_cells] - l31;//local_pattern[31];
                if(value!=0)break;
                value=patternArray_d[32 + mult_pos_tem_cells] - l32;//local_pattern[32];
                if(value!=0)break;
                value=patternArray_d[33 + mult_pos_tem_cells] - l33;//local_pattern[33];
                if(value!=0)break;
                value=patternArray_d[34 + mult_pos_tem_cells] - l34;//local_pattern[34];
                if(value!=0)break;
                value=patternArray_d[35 + mult_pos_tem_cells] - l35;//local_pattern[35];
                if(value!=0)break;
                value=patternArray_d[36 + mult_pos_tem_cells] - l36;//local_pattern[36];
                if(value!=0)break;
                value=patternArray_d[37 + mult_pos_tem_cells] - l37;//local_pattern[37];
                if(value!=0)break;
                value=patternArray_d[38 + mult_pos_tem_cells] - l38;//local_pattern[38];
                if(value!=0)break;
                value=patternArray_d[39 + mult_pos_tem_cells] - l39;//local_pattern[39];
                if(value!=0)break;
                value=patternArray_d[40 + mult_pos_tem_cells] - local_pattern[40];
                if(value!=0)break;
                value=patternArray_d[41 + mult_pos_tem_cells] - local_pattern[41];
                if(value!=0)break;
                value=patternArray_d[42 + mult_pos_tem_cells] - local_pattern[42];
                if(value!=0)break;
                value=patternArray_d[43 + mult_pos_tem_cells] - local_pattern[43];
                if(value!=0)break;
                value=patternArray_d[44 + mult_pos_tem_cells] - local_pattern[44];
                if(value!=0)break;
                value=patternArray_d[45 + mult_pos_tem_cells] - local_pattern[45];
                if(value!=0)break;
                value=patternArray_d[46 + mult_pos_tem_cells] - local_pattern[46];
                if(value!=0)break;
                value=patternArray_d[47 + mult_pos_tem_cells] - local_pattern[47];
                if(value!=0)break;
                value=patternArray_d[48 + mult_pos_tem_cells] - local_pattern[48];
                if(value!=0)break;
                value=patternArray_d[49 + mult_pos_tem_cells] - local_pattern[49];
                if(value!=0)break;
                value=patternArray_d[50 + mult_pos_tem_cells] - local_pattern[50];
                if(value!=0)break;
                value=patternArray_d[51 + mult_pos_tem_cells] - local_pattern[51];
                if(value!=0)break;
                value=patternArray_d[52 + mult_pos_tem_cells] - local_pattern[52];
                if(value!=0)break;
                value=patternArray_d[53 + mult_pos_tem_cells] - local_pattern[53];
                if(value!=0)break;
                value=patternArray_d[54 + mult_pos_tem_cells] - local_pattern[54];
                if(value!=0)break;
                value=patternArray_d[55 + mult_pos_tem_cells] - local_pattern[55];
                if(value!=0)break;
                value=patternArray_d[56 + mult_pos_tem_cells] - local_pattern[56];
                if(value!=0)break;
                value=patternArray_d[57 + mult_pos_tem_cells] - local_pattern[57];
                if(value!=0)break;
                value=patternArray_d[58 + mult_pos_tem_cells] - local_pattern[58];
                if(value!=0)break;
                value=patternArray_d[59 + mult_pos_tem_cells] - local_pattern[59];
                if(value!=0)break;
                value=patternArray_d[60 + mult_pos_tem_cells] - local_pattern[60];
                if(value!=0)break;
                value=patternArray_d[61 + mult_pos_tem_cells] - local_pattern[61];
                if(value!=0)break;
                value=patternArray_d[62 + mult_pos_tem_cells] - local_pattern[62];
                if(value!=0)break;
                value=patternArray_d[63 + mult_pos_tem_cells] - local_pattern[63];
                if(value!=0)break;
                value=patternArray_d[64 + mult_pos_tem_cells] - local_pattern[64];
                if(value!=0)break;
                value=patternArray_d[65 + mult_pos_tem_cells] - local_pattern[65];
                if(value!=0)break;
                value=patternArray_d[66 + mult_pos_tem_cells] - local_pattern[66];
                if(value!=0)break;
                value=patternArray_d[67 + mult_pos_tem_cells] - local_pattern[67];
                if(value!=0)break;
                value=patternArray_d[68 + mult_pos_tem_cells] - local_pattern[68];
                if(value!=0)break;
                value=patternArray_d[69 + mult_pos_tem_cells] - local_pattern[69];
                if(value!=0)break;
                value=patternArray_d[70 + mult_pos_tem_cells] - local_pattern[70];
                if(value!=0)break;
                value=patternArray_d[71 + mult_pos_tem_cells] - local_pattern[71];
                if(value!=0)break;
                value=patternArray_d[72 + mult_pos_tem_cells] - local_pattern[72];
                if(value!=0)break;
                value=patternArray_d[73 + mult_pos_tem_cells] - local_pattern[73];
                if(value!=0)break;
                value=patternArray_d[74 + mult_pos_tem_cells] - local_pattern[74];
                if(value!=0)break;
                value=patternArray_d[75 + mult_pos_tem_cells] - local_pattern[75];
                if(value!=0)break;
                value=patternArray_d[76 + mult_pos_tem_cells] - local_pattern[76];
                if(value!=0)break;
                value=patternArray_d[77 + mult_pos_tem_cells] - local_pattern[77];
                if(value!=0)break;
                value=patternArray_d[78 + mult_pos_tem_cells] - local_pattern[78];
                if(value!=0)break;
                value=patternArray_d[79 + mult_pos_tem_cells] - local_pattern[79];
                if(value!=0)break;
                value=patternArray_d[80 + mult_pos_tem_cells] - local_pattern[80];
                if(value!=0)break;
                value=patternArray_d[81 + mult_pos_tem_cells] - local_pattern[81];
                if(value!=0)break;
                value=patternArray_d[82 + mult_pos_tem_cells] - local_pattern[82];
                if(value!=0)break;
                value=patternArray_d[83 + mult_pos_tem_cells] - local_pattern[83];
                if(value!=0)break;
                value=patternArray_d[84 + mult_pos_tem_cells] - local_pattern[84];
                if(value!=0)break;
                value=patternArray_d[85 + mult_pos_tem_cells] - local_pattern[85];
                if(value!=0)break;
                value=patternArray_d[86 + mult_pos_tem_cells] - local_pattern[86];
                if(value!=0)break;
                value=patternArray_d[87 + mult_pos_tem_cells] - local_pattern[87];
                if(value!=0)break;
                value=patternArray_d[88 + mult_pos_tem_cells] - local_pattern[88];
                if(value!=0)break;
                value=patternArray_d[89 + mult_pos_tem_cells] - local_pattern[89];
                if(value!=0)break;
                value=patternArray_d[90 + mult_pos_tem_cells] - local_pattern[90];
                if(value!=0)break;
                value=patternArray_d[91 + mult_pos_tem_cells] - local_pattern[91];
                if(value!=0)break;
                value=patternArray_d[92 + mult_pos_tem_cells] - local_pattern[92];
                if(value!=0)break;
                value=patternArray_d[93 + mult_pos_tem_cells] - local_pattern[93];
                if(value!=0)break;
                value=patternArray_d[94 + mult_pos_tem_cells] - local_pattern[94];
                if(value!=0)break;
                value=patternArray_d[95 + mult_pos_tem_cells] - local_pattern[95];
                if(value!=0)break;
                value=patternArray_d[96 + mult_pos_tem_cells] - local_pattern[96];
                if(value!=0)break;
                value=patternArray_d[97 + mult_pos_tem_cells] - local_pattern[97];
                if(value!=0)break;
                value=patternArray_d[98 + mult_pos_tem_cells] - local_pattern[98];
                if(value!=0)break;
                value=patternArray_d[99 + mult_pos_tem_cells] - local_pattern[99];
                if(value!=0)break;
                value=patternArray_d[100 + mult_pos_tem_cells] - local_pattern[100];
                if(value!=0)break;
                value=patternArray_d[101 + mult_pos_tem_cells] - local_pattern[101];
                if(value!=0)break;
                value=patternArray_d[102 + mult_pos_tem_cells] - local_pattern[102];
                if(value!=0)break;
                value=patternArray_d[103 + mult_pos_tem_cells] - local_pattern[103];
                if(value!=0)break;
                value=patternArray_d[104 + mult_pos_tem_cells] - local_pattern[104];
                if(value!=0)break;
                value=patternArray_d[105 + mult_pos_tem_cells] - local_pattern[105];
                if(value!=0)break;
                value=patternArray_d[106 + mult_pos_tem_cells] - local_pattern[106];
                if(value!=0)break;
                value=patternArray_d[107 + mult_pos_tem_cells] - local_pattern[107];
                if(value!=0)break;
                value=patternArray_d[108 + mult_pos_tem_cells] - local_pattern[108];
                if(value!=0)break;
                value=patternArray_d[109 + mult_pos_tem_cells] - local_pattern[109];
                if(value!=0)break;
                value=patternArray_d[110 + mult_pos_tem_cells] - local_pattern[110];
                if(value!=0)break;
                value=patternArray_d[111 + mult_pos_tem_cells] - local_pattern[111];
                if(value!=0)break;
                value=patternArray_d[112 + mult_pos_tem_cells] - local_pattern[112];
                if(value!=0)break;
                value=patternArray_d[113 + mult_pos_tem_cells] - local_pattern[113];
                if(value!=0)break;
                value=patternArray_d[114 + mult_pos_tem_cells] - local_pattern[114];
                if(value!=0)break;
                value=patternArray_d[115 + mult_pos_tem_cells] - local_pattern[115];
                if(value!=0)break;
                value=patternArray_d[116 + mult_pos_tem_cells] - local_pattern[116];
                if(value!=0)break;
                value=patternArray_d[117 + mult_pos_tem_cells] - local_pattern[117];
                if(value!=0)break;
                value=patternArray_d[118 + mult_pos_tem_cells] - local_pattern[118];
                if(value!=0)break;
                value=patternArray_d[119 + mult_pos_tem_cells] - local_pattern[119];
                if(value!=0)break;
                value=patternArray_d[120 + mult_pos_tem_cells] - local_pattern[120];
                if(value!=0)break;
                value=patternArray_d[121 + mult_pos_tem_cells] - local_pattern[121];
                if(value!=0)break;
                value=patternArray_d[122 + mult_pos_tem_cells] - local_pattern[122];
                if(value!=0)break;
                value=patternArray_d[123 + mult_pos_tem_cells] - local_pattern[123];
                if(value!=0)break;
                value=patternArray_d[124 + mult_pos_tem_cells] - local_pattern[124];
                if(value!=0)break;
                atomicSub(&freq_realization_d[pos],1);
                minn = maxx + 1;
                break;
            }

            if (value > 0){
                maxx = pos -1;
            }
            else if (value < 0){
                minn = pos + 1;
            }
        }
    }
}


//
//
//        }
//}

// function called from main fortran program
extern "C" int fitnessfunctioncudawrapper_	(int *slides,int *cols,int *rows,int *tem_cells,
                                 int *tem_slides,int *tem_cols,int *tem_rows,
                                 int *tem_cells_slides,int *tem_cells_cols,int *tem_cells_rows,
				 int *npatterns, int *freq_realization, int *patternArray,
				 int *realization_matrix, DT *value)
{
    hipSetDevice(0);

    int *realization_matrix_d;
    int *freq_realization_d;
    int *tem_cells_slides_d, *tem_cells_cols_d, *tem_cells_rows_d;
    int *patternArray_d;
//    int shared_mem_size;
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
//    shared_mem_size = *npatterns*sizeof(int);
    dim3 threads(4,4,4);
    dim3 blocks( (*rows-*tem_rows + threads.x -1)/threads.x,
                    (*cols-*tem_cols + threads.y -1)/threads.y,
                    (*slides-*tem_slides + threads.z -1)/(threads.z) );
//    printf("bdimz = %d\n", blocks.z);
    // Allocate memory on GPU

    hipMalloc( (void **)&realization_matrix_d, sizeof(int) * (*slides * *cols * *rows) );
//    cudaMemset(realization_matrix_d, 0, sizeof(int) * (*slides * *cols * *rows));
    Check_CUDA_Error("ERROR:");
    hipMalloc( (void **)&freq_realization_d, sizeof(int) * (*npatterns) );
//    cudaMemset(freq_realization_d, 0, sizeof(int) * (*npatterns) );
    Check_CUDA_Error("ERROR:");
    hipMalloc( (void **)&patternArray_d, sizeof(int) * (*npatterns* *tem_cells));
//    cudaMemset(patternArray_d , 0, sizeof(int) * (*npatterns* *tem_cells));
    Check_CUDA_Error("ERROR:");
    hipMalloc( (void **)&tem_cells_slides_d, sizeof(int) * (*tem_cells) );
//    cudaMemset(tem_cells_slides_d, 0, sizeof(int) * (*tem_cells));
    Check_CUDA_Error("ERROR:");
    hipMalloc( (void **)&tem_cells_cols_d, sizeof(int) * (*tem_cells) );
//    cudaMemset(tem_cells_cols_d, 0, sizeof(int)*(*tem_cells));
    Check_CUDA_Error("ERROR:");
    hipMalloc( (void **)&tem_cells_rows_d, sizeof(int) * (*tem_cells) );
//    cudaMemset(tem_cells_rows_d, 0, sizeof(int)*(*tem_cells));
    Check_CUDA_Error("ERROR:");

    // copy vectors from CPU to GPU
    hipMemcpy( realization_matrix_d, realization_matrix,
        sizeof(int) * (*slides * *cols * *rows), hipMemcpyHostToDevice );
    Check_CUDA_Error("realization matrix host to device");
    hipMemcpy( freq_realization_d,   freq_realization  ,
        sizeof(int) * (*npatterns), hipMemcpyHostToDevice );
    Check_CUDA_Error("freq host to device");
    hipMemcpy( patternArray_d,   patternArray  ,
        sizeof(int) * (*npatterns* *tem_cells), hipMemcpyHostToDevice );
    Check_CUDA_Error("patternArray host to device");
    hipMemcpy( tem_cells_slides_d,   tem_cells_slides  ,
        sizeof(int) * (*tem_cells), hipMemcpyHostToDevice );
    Check_CUDA_Error("tem_cells_slides host to device");
    hipMemcpy( tem_cells_cols_d,     tem_cells_cols    ,
        sizeof(int) * (*tem_cells), hipMemcpyHostToDevice );
    Check_CUDA_Error("tem_cells_cols host to device");
    hipMemcpy( tem_cells_rows_d,     tem_cells_rows    ,
        sizeof(int) * (*tem_cells), hipMemcpyHostToDevice );
    Check_CUDA_Error("tem_cells_rows host to device");

 //   printf("value-in=%f\n",*value);
    if(!OPTIMIZED_KERNEL){
        hipEventRecord(start, 0);
        fitnessKernel<<< blocks, threads>>>(
                    *slides,*cols,*rows,*tem_cells,
                    *tem_slides,*tem_cols,*tem_rows,
                     tem_cells_slides_d,tem_cells_cols_d,tem_cells_rows_d,
                    *npatterns, freq_realization_d,patternArray_d,
                     realization_matrix_d);
        hipDeviceSynchronize();
        Check_CUDA_Error("fitness kernel");
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
//        printf ("Time for the not Optimized kernel: %f ms\n", time);
        printf ("%f\n", time/1000);
    }
    else{
        hipEventRecord(start, 0);
        fitnessKernelOptimized<<< blocks, threads>>>(
                    *slides,*cols,*rows,*tem_cells,
                    *tem_slides,*tem_cols,*tem_rows,
                     tem_cells_slides_d,tem_cells_cols_d,tem_cells_rows_d,
                    *npatterns, freq_realization_d,patternArray_d,
                     realization_matrix_d);
        hipDeviceSynchronize();
        Check_CUDA_Error("fitness kernel");
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
//        printf ("Time for the Optimized kernel: %f ms\n", time);
        printf ("%f\n", time/1000);
    }


    // copy vectors back from GPU to CPU
    hipMemcpy( freq_realization, freq_realization_d,
            sizeof(int) * (*npatterns ),  hipMemcpyDeviceToHost);
 //   Check_CUDA_Error("memcpy freq dev to host");

    *value=0.0;
    for(int i=0;i<*npatterns;i++)
        *value=*value + (DT)(freq_realization[i])*(DT)(freq_realization[i]);

//    printf("GPU value = %f\n",*value);

    // free device memory
    hipFree(realization_matrix_d);
    Check_CUDA_Error("ERROR:");
    hipFree(freq_realization_d  );
    Check_CUDA_Error("ERROR:");
    hipFree(patternArray_d      );
    Check_CUDA_Error("ERROR:");
    hipFree(tem_cells_slides_d  );
    Check_CUDA_Error("ERROR:");
    hipFree(tem_cells_cols_d    );
    Check_CUDA_Error("ERROR:");
    hipFree(tem_cells_rows_d    );
    Check_CUDA_Error("ERROR:");
    hipDeviceReset();
    return 0;
}
